#include "hip/hip_runtime.h"

#include "lib.h"

#include <hip/hip_complex.h>

__global__ void testKernel(uint16_t w, uint16_t h, uint32_t maxIter, double minRe, double maxRe, double minIm, double maxIm) {
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint16_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h)
    {
        double real = minRe + x * (maxRe - minRe) / w;
        double imag = minIm + y * (maxIm - minIm) / h;

        hipDoubleComplex constant = make_hipDoubleComplex(real, imag);
        hipDoubleComplex z = make_hipDoubleComplex(0, 0);

        uint16_t n = 0;
        for ( ; n < maxIter; ++n)
        {
            if (hipCabs(z) > 2.0)
                break;
            z = hipCadd(hipCmul(z, z), constant);
        }
    }
}



void BenchMark::benchGPU() {
    
    uint16_t WIDTH = 1920;
    uint16_t HEIGHT = 1080;
    uint32_t MAX_ITER = 500000;

    const double min_Re = -2.0;
    const double max_Re = 1.0;
    const double min_Im = -1.0;
    const double max_Im = 1.0;

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks( (WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                    (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y );
    
    auto start = std::chrono::high_resolution_clock().now();
    
    testKernel<<<threadsPerBlock, numBlocks>>>(
        WIDTH,
        HEIGHT,
        MAX_ITER,
        min_Re,
        max_Re,
        min_Im,
        max_Im
    );
    
    auto end = std::chrono::high_resolution_clock().now();
    std::chrono::duration<double> duration = end - start;
    
    printf("GPU - Fractal calculation took %f seconds.\n", duration.count());
        
    hipDeviceSynchronize();
}
